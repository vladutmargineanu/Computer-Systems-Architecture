#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "utils/utils.h"

#define NMAX		(1 << 20)

// ~TODO 3~
// Modify the kernel below such as each element of the 
// array will be now equal to 0 if it is an even number
// or 1, if it is an odd number
__global__ void kernel_parity_id(int *a, int N)
{
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N)
	{
		a[i] %= 2;
	}
}

// ~TODO 4~
// Modify the kernel below such as each element will
// be equal to the BLOCK ID this computation takes
// place.
__global__ void kernel_block_id(int *a, int N)
{
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N)
	{
		a[i] = blockIdx.x;
	}
}

// ~TODO 5~
// Modify the kernel below such as each element will
// be equal to the THREAD ID this computation takes
// place.
__global__ void kernel_thread_id(int *a, int N) 
{
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N)
	{
		a[i] = threadIdx.x;
	}
}

int main(void)
{
	int nDevices;
	hipDeviceProp_t prop;
	hipError_t err;
	int i;
	int* host_a;
	int* device_a;

	// Get the number of CUDA-capable GPU(s)
	hipGetDeviceCount(&nDevices);

	// ~TODO 1~
	// For each device, show some details in the format below, 
	// then set as active device the first one (assuming there
	// is at least CUDA-capable device). Pay attention to the
	// type of the fields in the hipDeviceProp_t structure.
	//
	// Device number: <i>
	//      Device name: <name>
	//      Total memory: <mem>
	//      Memory Clock Rate (KHz): <mcr>
	//      Memory Bus Width (bits): <mbw>
	// 
	// Hint: look for hipGetDeviceProperties and hipSetDevice in
	// the Cuda Toolkit Documentation. 
	for (i = 0; i < nDevices; ++i)
	{
		hipGetDeviceProperties(&prop, i);

		printf("Device number: %d\n", i);
		printf("\tDevice name: %s\n", prop.name);
		printf("\tTotal memory: %zu\n", prop.totalGlobalMem);
		printf("\tMemory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		printf("\tMemory Bus Width (bits): %d\n", prop.memoryBusWidth);
	}

	// ~TODO 2~
	// With information from example_2.cu, allocate an array with
	// integers (where a[i] = i). Then, modify the three kernels
	// above and execute them using 4 blocks, each with 4 threads.
	// Hint: num_elements = block_size * block_no (see example_2)
	//
	// You can use the fill_array_int(int *a, int n) function (from utils)
	// to fill your array as many times you want.
	host_a = (int*)malloc(NMAX * sizeof(*host_a));
	DIE(host_a == NULL, "malloc");

	err = hipMalloc(&device_a, NMAX * sizeof(*device_a));
	DIE(err != hipSuccess || device_a == NULL, "hipMalloc");

	fill_array_int(host_a, NMAX);

	err = hipMemcpy(device_a, host_a, NMAX * sizeof(*host_a),
		hipMemcpyHostToDevice);
	DIE(err != hipSuccess, "hipMemcpy");

	// ~TODO 3~
	// Execute kernel_parity_id kernel and then copy from 
	// the device to the host; call hipDeviceSynchronize()
	// after a kernel execution for safety purposes.
	//
	// Uncomment the line below to check your results
	kernel_parity_id<<<NMAX / 4, 4>>>(device_a, NMAX);

	err = hipDeviceSynchronize();
	DIE(err != hipSuccess, "hipDeviceSynchronize");

	err = hipMemcpy(host_a, device_a, NMAX * sizeof(*host_a),
		hipMemcpyDeviceToHost);
	DIE(err != hipSuccess, "hipMemcpy");

	check_task_1(3, host_a);

	// ~TODO 4~
	// Execute kernel_block_id kernel and then copy from 
	// the device to the host;
	//
	// Uncomment the line below to check your results
	kernel_block_id<<<NMAX / 4, 4>>>(device_a, NMAX);

	err = hipDeviceSynchronize();
	DIE(err != hipSuccess, "hipDeviceSynchronize");

	err = hipMemcpy(host_a, device_a, NMAX * sizeof(*host_a),
		hipMemcpyDeviceToHost);
	DIE(err != hipSuccess, "hipMemcpy");

	check_task_1(4, host_a);

	// ~TODO 5~
	// Execute kernel_thread_id kernel and then copy from 
	// the device to the host;
	//
	// Uncomment the line below to check your results
	kernel_thread_id<<<NMAX / 4, 4>>>(device_a, NMAX);

	err = hipDeviceSynchronize();
	DIE(err != hipSuccess, "hipDeviceSynchronize");

	err = hipMemcpy(host_a, device_a, NMAX * sizeof(*host_a),
		hipMemcpyDeviceToHost);
	DIE(err != hipSuccess, "hipMemcpy");

	check_task_1(5, host_a);

	// TODO 6: Free the memory
	free(host_a);
	err = hipFree(device_a);
	DIE(err != hipSuccess, "hipFree");
	
	return 0;
}
