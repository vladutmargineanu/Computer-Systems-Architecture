#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

// TODO 6: Write the code to add the two arrays element by element and 
// store the result in another array
__global__ void add_arrays(const float *a, const float *b, float *c, int N)
{
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N)
	{
		c[i] = a[i] + b[i];
	}
}

int main(void)
{
	hipSetDevice(0);
	int N = 1 << 20;
	hipError_t err;

	const size_t block_size = 256;
	size_t num_blocks;

	float *host_array_a = 0;
	float *host_array_b = 0;
	float *host_array_c = 0;

	float *device_array_a = 0;
	float *device_array_b = 0;
	float *device_array_c = 0;

	// TODO 1: Allocate the host's arrays
	// TODO 3: Check for allocation errors
	host_array_a = (float*)malloc(N * sizeof(*host_array_a));
	DIE(host_array_a == NULL, "malloc(host_array_a)");

	host_array_b = (float*)malloc(N * sizeof(*host_array_b));
	DIE(host_array_b == NULL, "malloc(host_array_b)");

	host_array_c = (float*)malloc(N * sizeof(*host_array_c));
	DIE(host_array_c == NULL, "malloc(host_array_c)");

	// TODO 2: Allocate the device's arrays
	err = hipMalloc(&device_array_a, N * sizeof(*device_array_a));
	DIE(err != hipSuccess || device_array_a == NULL,
		"hipMalloc(device_array_a)");

	err = hipMalloc(&device_array_b, N * sizeof(*device_array_b));
	DIE(err != hipSuccess || device_array_b == NULL,
		"hipMalloc(device_array_b)");

	err = hipMalloc(&device_array_c, N * sizeof(*device_array_c));
	DIE(err != hipSuccess || device_array_c == NULL,
		"hipMalloc(device_array_c)");

	// TODO 4: Fill array with values; use fill_array_float to fill
	// host_array_a and fill_array_random to fill host_array_b. Each
	// function has the signature (float *a, int n), where n = number of elements.
	fill_array_float(host_array_a, N);
	fill_array_random(host_array_b, N);

	// TODO 5: Copy the host's arrays to device
	err = hipMemcpy(device_array_a, host_array_a,
		N * sizeof(*host_array_a), hipMemcpyHostToDevice);
	DIE(err != hipSuccess, "hipMemcpy(host_array_a)");

	err = hipMemcpy(device_array_b, host_array_b,
		N * sizeof(*host_array_b), hipMemcpyHostToDevice);
	DIE(err != hipSuccess, "hipMemcpy(host_array_b)");

	// TODO 6: Execute the kernel, calculating first the grid size
	// and the amount of threads in each block from the grid
	// Hint: For this execise the block_size can have any value lower than the
	//      API's maximum value (it's recommended to be close to the maximum
	//      value).
	num_blocks = N / block_size;

	if (N % block_size)
	{
		++num_blocks;
	}

	add_arrays<<<num_blocks, block_size>>>(device_array_a, device_array_b,
		device_array_c, N);

	err = hipDeviceSynchronize();
	DIE(err != hipSuccess, "hipDeviceSynchronize");

	// TODO 7: Copy back the results and then uncomment the checking function
	err = hipMemcpy(host_array_c, device_array_c,
		N * sizeof(*host_array_c), hipMemcpyDeviceToHost);
	DIE(err != hipSuccess, "hipMemcpy(device_array_c)");

	check_task_2(host_array_a, host_array_b, host_array_c, N);

	// TODO 8: Free the memory
	free(host_array_a);
	free(host_array_b);
	free(host_array_c);

	err = hipFree(device_array_a);
	DIE(err != hipSuccess, "hipFree(device_array_a)");

	err = hipFree(device_array_b);
	DIE(err != hipSuccess, "hipFree(device_array_b)");

	err = hipFree(device_array_c);
	DIE(err != hipSuccess, "hipFree(device_array_c)");
   
	return 0;
}
