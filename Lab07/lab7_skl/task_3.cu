#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

#define BUF_2M		(2 * 1024 * 1024)
#define BUF_32M		(32 * 1024 * 1024)

int main(void)
{
	hipSetDevice(0);

	int *host_array_a = 0;
	int *host_array_b = 0;

	int *device_array_a = 0;
	int *device_array_b = 0;
	int *device_array_c = 0;

	hipError_t err;
	int i;

	// TODO 1: Allocate the host's arrays with the specified number of elements:
	// host_array_a => 32M
	// host_array_b => 32M
	host_array_a = (int*)malloc(BUF_32M * sizeof(*host_array_a));
	host_array_b = (int*)malloc(BUF_32M * sizeof(*host_array_b));

	// TODO 2: Allocate the device's arrays with the specified number of elements:
	// device_array_a => 32M
	// device_array_b => 32M
	// device_array_c => 2M
	hipMalloc(&device_array_a, BUF_32M * sizeof(*device_array_a));
	hipMalloc(&device_array_b, BUF_32M * sizeof(*device_array_b));
	hipMalloc(&device_array_c, BUF_2M * sizeof(*device_array_c));

	// Check for allocation errors
	if (host_array_a == 0 || host_array_b == 0 || 
		device_array_a == 0 || device_array_b == 0 || 
		device_array_c == 0)
	{
		printf("[*] Error!\n");
		return 1;
	}

	for (int i = 0; i < BUF_32M; ++i)
	{
		host_array_a[i] = i % 32;
		host_array_b[i] = i % 2;
	}

	printf("Before swap:\n");
	printf("a[i]\tb[i]\n");
	for (int i = 0; i < 10; ++i)
	{
		printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
	}

	// TODO 3: Copy from host to device
	err = hipMemcpy(device_array_a, host_array_a,
		BUF_32M * sizeof(*device_array_a), hipMemcpyHostToDevice);
	DIE(err != hipSuccess, "hipMemcpy(host_array_a)");

	err = hipMemcpy(device_array_b, host_array_b,
		BUF_32M * sizeof(*device_array_b), hipMemcpyHostToDevice);
	DIE(err != hipSuccess, "hipMemcpy(host_array_b)");

	// TODO 4: Swap the buffers (BUF_2M values each iteration)
	// Hint 1: device_array_c should be used as a temporary buffer
	// Hint 2: hipMemcpy
	for (i = 0; i != BUF_32M; i += BUF_2M)
	{
		err = hipMemcpy(device_array_c, device_array_b + i,
			BUF_2M * sizeof(*device_array_b),
			hipMemcpyDeviceToDevice);
		DIE(err != hipSuccess, "hipMemcpy(device_array_b)");

		err = hipMemcpy(device_array_b + i, device_array_a + i,
			BUF_2M * sizeof(*device_array_a),
			hipMemcpyDeviceToDevice);
		DIE(err != hipSuccess, "hipMemcpy(device_array_a)");

		err = hipMemcpy(device_array_a + i, device_array_c,
			BUF_2M * sizeof(*device_array_c),
			hipMemcpyDeviceToDevice);
		DIE(err != hipSuccess, "hipMemcpy(device_array_c)");
	}

	// TODO 5: Copy from device to host
	err = hipMemcpy(host_array_a, device_array_a,
		BUF_32M * sizeof(*host_array_a), hipMemcpyDeviceToHost);
	DIE(err != hipSuccess, "hipMemcpy(host_array_a)");

	err = hipMemcpy(host_array_b, device_array_b,
		BUF_32M * sizeof(*host_array_b), hipMemcpyDeviceToHost);
	DIE(err != hipSuccess, "hipMemcpy(host_array_b)");

	printf("\nAfter swap:\n");
	printf("a[i]\tb[i]\n");
	for (int i = 0; i < 10; ++i) {
		printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
	}

	// TODO 6: Free the memory
	free(host_array_a);
	free(host_array_b);

	err = hipFree(device_array_a);
	DIE(err != hipSuccess, "hipFree(device_array_a)");

	err = hipFree(device_array_b);
	DIE(err != hipSuccess, "hipFree(device_array_b)");

	err = hipFree(device_array_c);
	DIE(err != hipSuccess, "hipFree(device_array_c)");

	return 0;
}
