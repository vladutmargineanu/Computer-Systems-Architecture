

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>
#include <iostream>

#define TILE_WIDTH 16

// Task 1 - simple matrix multiplication
__global__ void matrix_multiply_simple(float *ma, float *mb, float *mc, size_t width)
{
	//TODO: calculate the row & column index of the element
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int i;

	float sum = 0;
	//TODO: do dot product between row of ma and column of mb
	for(i = 0; i < width; i++){
		sum = sum + ma[row * width + i] * mb[i * width + col];
	}

	//TODO: write result in mc
	mc[row * width + col] = sum;
}

// Task 2 - optimized matrix multiplication
__global__ void matrix_multiply(float *ma, float *mb, float *mc, size_t width)
{
	int tx = threadIdx.x, ty = threadIdx.y;
	int bx = blockIdx.x,  by = blockIdx.y;

	//TODO: allocate 2D tiles in __shared__ memory
	__shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
	__shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

	//TODO: calculate the row & column index of the element

	// loop over the tiles of the input
	float sum = 0;
	for(int t = 0; t < width/TILE_WIDTH; ++t) {
	
		//TODO: load tiles into __shared__ memory allocated before
		tileA[ty][tx] = ma[t + width * ty + tx];
		tileB[ty][tx] = mb[t + width * ty + tx];

		//TODO:
		// wait until all data is loaded before allowing
		// any thread in this block to continue
		__syncthreads();

		//TODO: do dot product between row of tile from ma and column of tile from mb
		for(int i = 0; i < TILE_WIDTH; i++){
			sum = sum + tileA[ty][i] * tileB[i][tx];
		}

		//TODO:
		// wait until all data is loaded before allowing
		// any thread in this block to continue
		__syncthreads();
	}

	mc[width * TILE_WIDTH * by + TILE_WIDTH * bx + width * ty + tx] = sum;
	//TODO: write result in mc
}

int main(void)
{
	// create a large workload so we can easily measure the
	// performance difference of both implementations

	// note that n measures the width of the matrix, not the number of total elements
	const size_t n = 1<<10;
	const dim3 block_size(TILE_WIDTH,TILE_WIDTH);
	const dim3 num_blocks(n / block_size.x, n / block_size.y);

	// generate random input on the host
	std::vector<float> host_a(n*n), host_b(n*n), host_c(n*n);
	for(int i = 0; i < n*n; ++i) {
		host_a[i] = static_cast<float>(rand()) / RAND_MAX;
		host_b[i] = static_cast<float>(rand()) / RAND_MAX;
	}

	// allocate storage for the device
	float *device_a = 0, *device_b = 0, *device_c = 0;
	hipMalloc((void**)&device_a, sizeof(float) * n * n);
	hipMalloc((void**)&device_b, sizeof(float) * n * n);
	hipMalloc((void**)&device_c, sizeof(float) * n * n);

	// copy input to the device
	hipMemcpy(device_a, &host_a[0], sizeof(float) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(device_b, &host_b[0], sizeof(float) * n * n, hipMemcpyHostToDevice);

	//Task 3 - measure the time spent in the kernel for simple and optimized implementation
	
	//TODO: create CUDA events for measuring kernel time
	hipEvent_t launch_begin, launch_end;

	// time many kernel launches and take the average time
	const size_t num_launches = 100;
	float average_simple_time = 0;
	std::cout << "Timing simple implementation...";
	hipEventCreate(&launch_begin);
	hipEventCreate(&launch_end);

	for(int i = 0; i < num_launches; ++i) {	
		//TODO: record CUDA event before and after the kernel launch
		hipEventRecord(launch_begin);
		matrix_multiply_simple<<<num_blocks,block_size>>>(device_a, device_b, device_c, n);
		//TODO: Wait for launch_end event to complete
		hipEventRecord(launch_end);
		hipEventSynchronize(launch_end);
		
		//TODO: measure the time spent in the kernel
		float time = 0;
		hipEventElapsedTime(&time, launch_begin, launch_end);
		average_simple_time += time;
	}
	average_simple_time /= num_launches;
	std::cout << " done." << std::endl;

	//now time the optimized kernel

	// time many kernel launches and take the average time
	float average_optimized_time = 0;
	std::cout << "Timing optimized implementation...";
	for(int i = 0; i < num_launches; ++i) {
		//TODO: record CUDA event before and after the kernel launch
		hipEventRecord(launch_begin);
		matrix_multiply<<<num_blocks,block_size>>>(device_a, device_b, device_c, n);

		//TODO: Wait for launch_end event to complete
		hipEventRecord(launch_end);
		hipEventSynchronize(launch_end);
		
		//TODO: measure the time spent in the kernel
		float time = 0;
		hipEventElapsedTime(&time, launch_begin, launch_end);
		average_optimized_time += time;
	}
	average_optimized_time /= num_launches;
	std::cout << " done." << std::endl;

	// report the effective throughput of each kernel in GFLOPS
	// the effective throughput is measured as the number of floating point operations performed per second:
	// (one mul + one add) * N^3
	float simple_throughput = static_cast<float>(2 * n * n * n) / (average_simple_time / 1000.0f) / 1000000000.0f;
	float optimized_throughput = static_cast<float>(2 * n * n * n) / (average_optimized_time / 1000.0f) / 1000000000.0f;

	std::cout << "Matrix size: " << n << "x" << n << std::endl;
	std::cout << "Tile size: " << TILE_WIDTH << "x" << TILE_WIDTH << std::endl;

	std::cout << "Throughput of simple kernel: " << simple_throughput << " GFLOPS" << std::endl;
	std::cout << "Throughput of optimized kernel: " << optimized_throughput << " GFLOPS" << std::endl;
	std::cout << "Performance improvement: " << optimized_throughput / simple_throughput << "x" << std::endl;
	std::cout << std::endl;
	
	//TODO: destroy the CUDA events
	hipEventDestroy(launch_begin);
	hipEventDestroy(launch_end);

	// deallocate device memory
	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_c);

	return 0;
}

